#include <stdio.h>
#include <hip/hip_runtime.h>
#define w  358
#define h 1	//colonne
#define COl 5248
#define ROW 358
#define N w*h

__global__ void reduce(float*g_idata, float*g_odata);
void fill_array (float*a, int n);
void stampa_mat(float*a);

int main( void ) {
	printf("Entrato nel main\n");
    float a[N], b[N];
    float*dev_a, *dev_b;
    int size = N * sizeof( float); // we need space for N integers

    // allocate device copies of a, b, c
    hipMalloc( (void**)&dev_a, size );
    hipMalloc( (void**)&dev_b, size );

    fill_array( a, N );
//    printf("Valori che voglio ottenere dalla riduzione\n");
    printf("+-------------------------+\n");
    //stampa_mat(a);
    printf("+-------------------------+\n\n");
    for (int i = 0; i < h; ++i) {//colonna
    	float tot = 0;
    	for (int j = 0;  j < w; j++) {//riga
    		tot += a[j*h+i];
		}
    	printf("tot[%d] = %lf\n",i,tot);
	}
    // copy inputs to device
    hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice );

    dim3 blocksize(w); // create 1D threadblock
    dim3 gridsize(h);  //create 1D grid

    //128*5248
    reduce<<<gridsize, blocksize>>>(dev_a, dev_b);

    // copy device result back to host copy of c
    hipMemcpy( b, dev_b, size, hipMemcpyDeviceToHost );
    printf("Reduced sum of Array elements = %lf \n", b[0]);
    printf("Value should be: %d \n", 28);
    hipFree( dev_a );
    hipFree( dev_b );

    return 0;
}

__global__ void reduce(float*g_idata, float*g_odata) {
    __shared__ float sdata[w]; // w=128

    // each thread loads one element from global to shared mem
    // note use of 1D thread indices (only) in this kernel
    int i = blockIdx.x + gridDim.x*threadIdx.x; //gridDIm = h, blockDIm = w
    unsigned int tid = threadIdx.x;
    //blockDim = 256;
    //blockIdx.x = 0,1,2,3;
    if(blockIdx.x == 0)
    	printf("data = %lf\n", g_idata[i]);

    sdata[tid] = g_idata[i]; //sdata filled with g_idata per block
	//printf("[loaded]	sdata[%f]=%f, blckIdx.x:%f\n",i,sdata[threadIdx.x],blockIdx.x);
    __syncthreads();
    // do reduction in shared mem
    for (int s=1; s < blockDim.x; s *=2)
    {
        int index = 2 * s * threadIdx.x;
        if (index < blockDim.x ){
        	if(index+s >= blockDim.x)
        		sdata[index+s] = 0;
        	sdata[index] += sdata[index + s];
        }
    	__syncthreads();
    }
    if (tid == 0){
    	//printf("[reduction]	sdata[%d]=%f\n",i,sdata[tid]);
        if(blockIdx.x == 0)
        	printf("red_tot = %lf\n", sdata[tid]);
    	g_odata[blockIdx.x] = sdata[tid]/ROW;
    		__syncthreads();
		}

//        atomicAdd(g_odata,sdata[0]); //prende tutti i valori dei vari blocchi e li somma
        /*
         * in questo caso quel che succede è:
         * sdata[0]=256; //blc0
         * sdata[0]=256; //blc1
         * sdata[0]=256; //blc2
         * sdata[0]=256; //blc3
         * L'atomic add restituisce 1024
         */

}

// CPU function to generate a vector of random integers
void fill_array (float*a, int n)
{
    for (int i = 0; i < n; i++)
        a[i] = 1;
}
void stampa_mat(float*a){
	int i, j;
	for (i = 0; i < w; ++i) {
		for (j = 0; j < h; ++j) {
			printf("%f\t",a[j+i*h]);
		}
		printf("\n");
	}
}
